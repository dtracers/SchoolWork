//
// CUDA code to compute minimu distance between n points
//

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define MAX_POINTS 1048576
#define BLOCK_SIZE 1024

// ----------------------------------------------------------------------------
// Kernel Function to compute distance between all pairs of points
// Input:
//      X: X[i] = x-coordinate of the ith point
//      Y: Y[i] = y-coordinate of the ith point
//      n: number of points
//
// Output:
//      D: D[0] = minimum distance
//
__global__ void minimum_distance(float * X, float * Y, float * D, float * Glob,
		int n) {
	unsigned int i = ((blockIdx.x * blockDim.x) + threadIdx.x);
	int j = 0;
	if (i < n) {
		// complete first stop to initalize!!!!!!
		float xFirst = X[i], yFirst = Y[i];
		float xComp = X[i + 1], yComp = Y[i + 1];
		float sqX = (xComp - xFirst);
		float sqY = (yComp - yFirst);
		D[i] = sqrtf(sqX * sqX + sqY * sqY);

		for (j = i + 1; j < n; j++) {
			xComp = X[j];
			yComp = Y[j];
			float sqX = (xComp - xFirst);
			float sqY = (yComp - yFirst);
			float distance = sqrtf(sqX * sqX + sqY * sqY);
			if (distance < D[i])
				D[i] = distance;
		}

	}
	__syncthreads();
	if (i < blockDim.x && i < n) {
		float sdata;
		int tid = threadIdx.x;
		int shift = n;
		if (blockDim.x < n) {
			shift = n / blockDim.x;
			sdata = D[tid * shift];
			for (j = 1; j < shift; j++) {
				if (sdata > D[(tid * shift) + j])
					sdata = D[(tid * shift) + j];
			}
			shift = blockDim.x;
			D[tid] = sdata;
		}

		__syncthreads();

		for (unsigned int s = shift / 2; s > 0; s >>= 1) {
			if (tid < s) {
				if (D[tid] > D[tid + s]) {
					D[tid] = D[tid + s];
				}
			}
			__syncthreads();
		}

		if (i == 0)
			*Glob = D[0];
	}

}
// ----------------------------------------------------------------------------
// Main program - initializes points and computes minimum distance
// between the points
//
int main(int argc, char* argv[]) {

	// Host Data
	float * hVx;                // host x-coordinate array
	float * hVy;                // host y-coordinate array
	float * hmin_dist;          // minimum value on host

	// Device Data
	float * dVx;                // device x-coordinate array
	float * dVy;                // device x-coordinate array
	float * dmin_dist;          // minimum value on device
	float * dVd;

	int i, j, size, num_points, threads, blocks;
	float dx, dy, Dij, distance;
	unsigned int seed = 0;

	hipEvent_t start, stop;            // GPU timing variables

	struct timeval cpu_start, cpu_stop; // CPU timing variables

	float time_array[10];

	// Timing initializations
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Check input
	if (argc != 2) {
		printf("Use: %s <number of points>\n", argv[0]);
		exit(0);
	}
	if ((num_points = atoi(argv[argc - 1])) > MAX_POINTS) {
		printf("Maximum number of points allowed: %d\n", MAX_POINTS);
		exit(0);
	}

	// Allocate host coordinate arrays
	size = num_points * sizeof(float);
	hVx = (float *) malloc(size);
	hVy = (float *) malloc(size);
	hmin_dist = (float *) malloc(sizeof(float));

	// Initialize points
	for (i = 0; i < num_points; i++) {
		hVx[i] = (float) (rand_r(&seed)) / (float) (RAND_MAX);
		hVy[i] = (float) (rand_r(&seed)) / (float) (RAND_MAX);
	}

	// Allocate device coordinate arrays
	hipMalloc(&dVx, size);
	hipMalloc(&dVy, size);
	hipMalloc(&dmin_dist, sizeof(float));
	hipMalloc(&dVd, size);

	// Copy coordinate arrays from host memory to device memory
	hipEventRecord(start, 0);

	hipMemcpy(dVx, hVx, size, hipMemcpyHostToDevice);
	hipMemcpy(dVy, hVy, size, hipMemcpyHostToDevice);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&(time_array[0]), start, stop);

	// Invoke kernel
	hipEventRecord(start, 0);

	threads = 256;
	blocks = 1;
	if (num_points > threads) {
		blocks = num_points / threads;
	}

	minimum_distance<<<blocks,threads>>>(dVx, dVy, dVd, dmin_dist, num_points);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&(time_array[1]), start, stop);

	// Copy result from device memory to host memory
	hipEventRecord(start, 0);

	hipMemcpy(hmin_dist, dmin_dist, sizeof(float), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&(time_array[2]), start, stop);

	printf("Number of Points = %d\n", num_points);
	printf("GPU Host-to-device = %f ms \n", time_array[0]);
	printf("GPU execution time = %f ms \n", time_array[1]);
	printf("GPU Device-to-host = %f ms \n", time_array[2]);
	printf("Minimum distance (GPU) = %e\n", hmin_dist[0]);

	// Compute minimum distance on host to check device computation
	gettimeofday(&cpu_start, NULL);
	dx = hVx[1] - hVx[0];
	dy = hVy[1] - hVy[0];
	distance = sqrtf(dx * dx + dy * dy);
	for (i = 0; i < num_points; i++) {
		for (j = i + 1; j < num_points; j++) {
			dx = hVx[j] - hVx[i];
			dy = hVy[j] - hVy[i];
			Dij = sqrtf(dx * dx + dy * dy);
			if (distance > Dij)
				distance = Dij;
		}
	}
	gettimeofday(&cpu_stop, NULL);
	time_array[3] = 1000 * (cpu_stop.tv_sec - cpu_start.tv_sec)
			+ 0.000001 * (cpu_stop.tv_usec - cpu_start.tv_usec);
	printf("CPU execution time = %f ms\n", time_array[3]);
	printf("Minimum distance (CPU) = %e\n", distance);

	// Free device memory
	hipFree(dVx);
	hipFree(dVy);
	hipFree(dmin_dist);
	hipFree(dVd);

	// Free host memory
	free(hVx);
	free(hVy);
	free(hmin_dist);

	return 0;
}

