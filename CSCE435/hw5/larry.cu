//
// CUDA code to compute minimu distance between n points
//

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define MAX_POINTS 1048576
#define BLOCK_SIZE 1024

// ---------------------------------------------------------------------------- 
// Kernel Function to compute distance between all pairs of points
// Input: 
//	X: X[i] = x-coordinate of the ith point
//	Y: Y[i] = y-coordinate of the ith point
//	n: number of points
//
// Output: 
//	D: D[0] = minimum distance
//
__global__ void minimum_distance(float * X, float * Y, float * D, float * M,
		int n) {
	unsigned int i = ((blockIdx.x * blockDim.x) + threadIdx.x);
	int j = 0;
	if (i < n) {
		float x_main = X[i], y_main = Y[i];
		float x_comparison = X[i + 1], y_comparison = Y[i + 1];
		float squaredx = (x_comparison - x_main);
		float squaredy = (y_comparison - y_main);
		D[i] = sqrtf(squaredx * squaredx + squaredy * squaredy);

		for (j = i + 1; j < n; j++) {
			x_comparison = X[j];
			y_comparison = Y[j];
			float squaredx = (x_comparison - x_main);
			float squaredy = (y_comparison - y_main);
			float distance = sqrtf(squaredx * squaredx + squaredy * squaredy);
			if (distance < D[i])
				D[i] = distance;
		}
	}
	__syncthreads();
	if (i < blockDim.x) {
		int tid = threadIdx.x;
		int change = n / blockDim.x;
		float sdata = D[tid * change];
		for (j = 1; j < change; j++) {
			if (sdata > D[(tid * change) + j])
				sdata = D[(tid * change) + j];
		}
		__syncthreads();
		D[tid] = sdata;
		// do reduction in shared mem
		for (unsigned int s = 1; s < blockDim.x; s *= 2) {
			__syncthreads();
			if (tid % (2 * s) == 0) {
				if (D[tid] > D[tid + s])
					D[tid] = D[tid + s];
			}
			__syncthreads();
		}

		// write result for this block to global mem
		if (i == 0)
			*M = X[0];
	}

}
// ---------------------------------------------------------------------------- 
// Main program - initializes points and computes minimum distance 
// between the points
//
int main(int argc, char* argv[]) {

	// Host Data
	float * hVx;                // host x-coordinate array
	float * hVy;                // host y-coordinate array
	float * hmin_dist;          // minimum value on host

	// Device Data
	float * dVx;                // device x-coordinate array
	float * dVy;                // device x-coordinate array
	float * dmin_dist;          // minimum value on device
	float * dVd;

	int i, j, size, num_points;
	float dx, dy, Dij, distance;
	unsigned int seed = 0;

	hipEvent_t start, stop;            // GPU timing variables

	struct timeval cpu_start, cpu_stop; // CPU timing variables

	float time_array[10];

	// Timing initializations
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Check input
	if (argc != 2) {
		printf("Use: %s <number of points>\n", argv[0]);
		exit(0);
	}
	if ((num_points = atoi(argv[argc - 1])) > MAX_POINTS) {
		printf("Maximum number of points allowed: %d\n", MAX_POINTS);
		exit(0);
	}

	// Allocate host coordinate arrays
	size = num_points * sizeof(float);
	hVx = (float *) malloc(size);
	hVy = (float *) malloc(size);
	hmin_dist = (float *) malloc(sizeof(float));

	// Initialize points
	for (i = 0; i < num_points; i++) {
		hVx[i] = (float) (rand_r(&seed)) / (float) (RAND_MAX);
		hVy[i] = (float) (rand_r(&seed)) / (float) (RAND_MAX);
	}

	// Allocate device coordinate arrays
	hipMalloc(&dVx, size);
	hipMalloc(&dVy, size);
	hipMalloc(&dmin_dist, sizeof(float));
	//cudaMalloc(&dVd,size);
	// Copy coordinate arrays from host memory to device memory
	hipEventRecord(start, 0);

	hipMemcpy(dVx, hVx, size, hipMemcpyHostToDevice);
	hipMemcpy(dVy, hVy, size, hipMemcpyHostToDevice);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&(time_array[0]), start, stop);

	// Invoke kernel
	hipEventRecord(start, 0);

	minimum_distance<<<1, 16>>>(dVx, dVy, dVd, dmin_dist, num_points);

	// ------------------------------------------------------------
	/* Define block size */
	//threads = 16;
	//blocks = (num_points/threads);
	/* Invoke kernel */

	// ------------------------------------------------------------
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&(time_array[1]), start, stop);

	// Copy result from device memory to host memory
	hipEventRecord(start, 0);

	hipMemcpy(hmin_dist, dmin_dist, sizeof(float), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&(time_array[2]), start, stop);

	printf("Number of Points = %d\n", num_points);
	printf("GPU Host-to-device = %f ms \n", time_array[0]);
	printf("GPU execution time = %f ms \n", time_array[1]);
	printf("GPU Device-to-host = %f ms \n", time_array[2]);
	printf("Minimum distance (GPU) = %e\n", hmin_dist[0]);

	// Compute minimum distance on host to check device computation
	gettimeofday(&cpu_start, NULL);
	dx = hVx[1] - hVx[0];
	dy = hVy[1] - hVy[0];
	distance = sqrtf(dx * dx + dy * dy);
	for (i = 0; i < num_points; i++) {
		for (j = i + 1; j < num_points; j++) {
			dx = hVx[j] - hVx[i];
			dy = hVy[j] - hVy[i];
			Dij = sqrtf(dx * dx + dy * dy);
			if (distance > Dij)
				distance = Dij;
		}
	}
	gettimeofday(&cpu_stop, NULL);
	time_array[3] = 1000 * (cpu_stop.tv_sec - cpu_start.tv_sec)
			+ 0.000001 * (cpu_stop.tv_usec - cpu_start.tv_usec);
	printf("CPU execution time = %f ms\n", time_array[3]);
	printf("Minimum distance (CPU) = %e\n", distance);

	// Free device memory 
	hipFree(dVx);
	hipFree(dVy);
	hipFree(dmin_dist);

	// Free host memory 
	free(hVx);
	free(hVy);
	free(hmin_dist);

	return 0;
}
