//
// CUDA code to compute minimum distance between n points
//

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define MAX_POINTS 1048576
#define BLOCK_SIZE 1024
#define M_LOG2E 1.44269504088896340736 //log2(e)

inline long double my_log(const long double x){
    return  log(x) * M_LOG2E;
}

int blocks;     //log2(num_points);
int threads;    //num_points/blocks;
int num_points;

// ---------------------------------------------------------------------------- 
// Kernel Function to compute distance between all pairs of points
// Input: 
//  X: X[i] = x-coordinate of the ith point
//  Y: Y[i] = y-coordinate of the ith point
//  n: number of points
//
// Output: 
//  D: D[0] = minimum distance
// ----------------------------------------------------------------------------

// ----------------------------------------------------------------------------
// Compute minimum distance on host to check device computation
// Need to:  modify the kernel functino to compute the minimum distance.
// You are allowed to make other changes to the code that facilitate
// parallelization on the GPU
//
// 1. You need to develop CUDA-based parallel code to compute the
// distance between the closest pair of points on a GPU
// ----------------------------------------------------------------------------

// ----------------------------------------------------------------------------
// References
// ----------------------------------------------------------------------------
/* 
__global__ void VecAdd(float* A, float* B, float* C, int N) 
{ 
 int i = blockDim.x * blockIdx.x + threadIdx.x; 
 if (i < N) 
 C[i] = A[i] + B[i]; 
} 
*/

/*
__global__ void MatMult(float* A, float* B, float* C, int N) 
{ 
int i = blockIdx.x * blockDim.x + threadIdx.x; 
 int j = blockIdx.y * blockDim.y + threadIdx.y; 
 if (i < N && j < N) 
 C[i][j] = A[i][j] + B[i][j];
 }
 */
// ----------------------------------------------------------------------------

// ----------------------------------------------------------------------------
// Kernel function code goes here
// ---------------------------------------------------------------------------- 
__global__ void minimum_distance(float * X, float * Y, float * D, int n) 
{
    //CUDA version of the code
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    //int j = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if(threadIdx.y == 0)
    {
        D[0] = 10;
    }

    extern __shared__ float mins[]; 
    __threadfence();

    //if (i < n && j < n) 
    if(i<n)
    {
        for (int j = i+1; j < n; j++) 
        {
            float dx = X[j]-X[i];
            float dy = Y[j]-Y[i];
            float Dij = sqrtf(dx*dx+dy*dy);
            
            //__threadfence();

            //mins local to the block
            mins[threadIdx.x] = Dij;

            __syncthreads();
        }
    }

    //let the last block handle finding the actual minimum
    if(i >= n)
    {
        //sort mins to 
        for(unsigned int s = blockDim.y/2; s > 0; s >>= 1) 
        {

            if(threadIdx.y < s)
            {
                if (mins[threadIdx.y] > mins[threadIdx.y + s]) 
                {
                    mins[threadIdx.y] = mins[threadIdx.y + s];
                }
            }
            __syncthreads();
        }

        //now update global minimum
        if(threadIdx.y == 0)
        {
            if(D[0] > mins[0])
            {
                D[0] = mins[0];
            }
        }
    }
    //D[0] = -15;
} 
// ---------------------------------------------------------------------------- 

// ---------------------------------------------------------------------------- 
// Main program - initializes points and computes minimum distance 
// between the points
// ---------------------------------------------------------------------------- 
int main(int argc, char* argv[]) {

    // Host Data
    float * hVx;            // host x-coordinate array
    float * hVy;            // host y-coordinate array
    float * hmin_dist;      // minimum value on host

    // Device Data
    float * dVx;            // device x-coordinate array
    float * dVy;            // device x-coordinate array
    float * dmin_dist;      // minimum value on device

    int i, j, size;// num_points, blocks, threads; 
    float dx, dy, Dij, distance;
    unsigned int seed = 0;

    hipEvent_t start, stop;        // GPU timing variables

    struct timeval cpu_start, cpu_stop; // CPU timing variables

    float time_array[10]; 

    // Timing initializations
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Check input
    if (argc != 2) {
    printf("Use: %s <number of points>\n", argv[0]);  
    exit(0);
    }
    if ((num_points = atoi(argv[argc-1])) > MAX_POINTS) {
    printf("Maximum number of points allowed: %d\n", MAX_POINTS);
    exit(0);
    } 

    // Allocate host coordinate arrays 
    size = num_points * sizeof(float); 
    hVx = (float *) malloc(size); 
    hVy = (float *) malloc(size);
    hmin_dist = (float *) malloc(sizeof(float));

    // Initialize points
    for (i = 0; i < num_points; i++) {
        hVx[i] = (float)(rand_r(&seed))/(float)(RAND_MAX);
        hVy[i] = (float)(rand_r(&seed))/(float)(RAND_MAX);
    }

    // Allocate device coordinate arrays
    hipMalloc(&dVx, size);
    hipMalloc(&dVy, size);
    hipMalloc(&dmin_dist, sizeof(float));

    // Copy coordinate arrays from host memory to device memory 
    hipEventRecord( start, 0 ); 

    hipMemcpy(dVx, hVx, size, hipMemcpyHostToDevice);
    hipMemcpy(dVy, hVy, size, hipMemcpyHostToDevice);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&(time_array[0]), start, stop);

    // Invoke kernel
    hipEventRecord(start, 0); 

    // ------------------------------------------------------------
    //
    // Invoke kernel function here
    //
    // ------------------------------------------------------------
    //choose the number of blocks and points
    threads = 16;                      //num_points/blocks;
    blocks = num_points/threads;       //my_log(num_points);
    minimum_distance<<<(blocks), threads>>>(dVx, dVy, dmin_dist, num_points);

    //alternative declaration
    //dim3 block(16,16);
    //dim3 grid((num_points + 15)/16, (num_points + 15)/16);
    //minimum_distance<<<grid, block>>>(dVx, dVy, dmin_dist, num_points);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&(time_array[1]), start, stop);

    // Copy result from device memory to host memory 
    hipEventRecord(start, 0); 

    //this was wrong! converting size to sizeof(float)
    hipMemcpy(hmin_dist, dmin_dist, sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&(time_array[2]), start, stop);

    printf("Number of Points = %d\n", num_points); 
    printf("GPU Host-to-device = %f ms \n", time_array[0]);
    printf("GPU execution time = %f ms \n", time_array[1]);
    printf("GPU Device-to-host = %f ms \n", time_array[2]);
    printf("Minimum distance (GPU) = %e\n", hmin_dist[0]);

    gettimeofday(&cpu_start, NULL);

    dx = hVx[1]-hVx[0];
    dy = hVy[1]-hVy[0];
    distance = sqrtf(dx*dx+dy*dy);
    
    for (i = 0; i < num_points; i++) {
       for (j = i+1; j < num_points; j++) {
           dx = hVx[j]-hVx[i];
           dy = hVy[j]-hVy[i];
           Dij = sqrtf(dx*dx+dy*dy);
           if (distance > Dij) distance = Dij;
       }
    }

    gettimeofday(&cpu_stop, NULL); 
    time_array[3] = 1000*(cpu_stop.tv_sec-cpu_start.tv_sec)
    +0.000001*(cpu_stop.tv_usec-cpu_start.tv_usec);
    printf("CPU execution time = %f ms\n", time_array[3]);
    printf("Minimum distance (CPU) = %e\n", distance);

    // Free device memory 
    hipFree(dVx);
    hipFree(dVy);
    hipFree(dmin_dist);

    // Free host memory 
    free(hVx);
    free(hVy);
    free(hmin_dist);

    return 0;
}  
// ---------------------------------------------------------------------------- 
