//
// CUDA code to compute minimu distance between n points
//

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define MAX_POINTS 1048576
#define BLOCK_SIZE 1024

int blocks;     //log2(num_points);
int threads;    //num_points/blocks;
int num_points;

//__device__ float mins[BLOCK_SIZE];

// ---------------------------------------------------------------------------- 
// Kernel Function to compute distance between all pairs of points
// Input: 
//  X: X[i] = x-coordinate of the ith point
//  Y: Y[i] = y-coordinate of the ith point
//  n: number of points
//
// Output: 
//  D: D[0] = minimum distance
//
__global__ void minimum_distance(float * X, float * Y, float * D, int n) {
	//CUDA version of the code
	//D[0] = (float)blockDim.x;
	//return;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//int j = blockIdx.x * blockDim.x + threadIdx.x +1;

	//int i = threadIdx.x;
	//int j = threadIdx.x + 1;

	if (i == 0) {
		D[0] = 999;
	}

	extern __shared__ float mins[];
	__threadfence();

	float distance = 999;
	//D[0] = distance;
	//printf((int)distance);

	if (i < n) {
		for (int j = i + 1; j < n; j++) {
			float dx = X[j] - X[i];
			float dy = Y[j] - Y[i];
			float Dij = sqrtf(dx * dx + dy * dy);
			if (distance > Dij) {
				distance = Dij;
			}
		}

		//mins local to the block
		mins[threadIdx.x] = distance;
	}
	//return;

	__syncthreads();

	if (i < blockDim.x) {
		int maxValue = blockDim.x;
		if (maxValue > n) {
			maxValue = n;
		}
		int index = threadIdx.x;
		for (unsigned int s = maxValue / 2; s > 0; s >>= 1) {
			if (index < s) {
				if (mins[index] > mins[index + s - 1]
						&& mins[index + s - 1] != 0)
					mins[index] = mins[index + s - 1];
			}
			__syncthreads();
		}
		if (i == 0) {
			if (D[0] > mins[0] && mins[0] != 0) {
				D[0] = mins[0];
			}
		}

	}
//    return;

	//now update global minimum

}

// ---------------------------------------------------------------------------- 
// Main program - initializes points and computes minimum distance 
// between the points
//
int main(int argc, char* argv[]) {

	// Host Data
	float * hVx;    // host x-coordinate array
	float * hVy;    // host y-coordinate array
	float * hmin_dist;    // minimum value on host

	// Device Data
	float * dVx;    // device x-coordinate array
	float * dVy;    // device x-coordinate array
	float * dmin_dist;    // minimum value on device

	int i, j, size;    //, num_points, blocks, threads; 
	float dx, dy, Dij, distance;
	unsigned int seed = 0;

	hipEvent_t start, stop;    // GPU timing variables

	struct timeval cpu_start, cpu_stop;    // CPU timing variables

	float time_array[10];

	// Timing initializations
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Check input
	if (argc != 2) {
		printf("Use: %s <number of points>\n", argv[0]);
		exit(0);
	}
	if ((num_points = atoi(argv[argc - 1])) > MAX_POINTS) {
		printf("Maximum number of points allowed: %d\n", MAX_POINTS);
		exit(0);
	}

	// Allocate host coordinate arrays 
	size = num_points * sizeof(float);
	hVx = (float *) malloc(size);
	hVy = (float *) malloc(size);
	hmin_dist = (float *) malloc(size);

	// Initialize points
	for (i = 0; i < num_points; i++) {
		hVx[i] = (float) (rand_r(&seed)) / (float) (RAND_MAX);
		hVy[i] = (float) (rand_r(&seed)) / (float) (RAND_MAX);
	}

	// Allocate device coordinate arrays
	hipMalloc(&dVx, size);
	hipMalloc(&dVy, size);
	hipMalloc(&dmin_dist, sizeof(float));

	// Copy coordinate arrays from host memory to device memory 
	hipEventRecord(start, 0);

	hipMemcpy(dVx, hVx, size, hipMemcpyHostToDevice);
	hipMemcpy(dVy, hVy, size, hipMemcpyHostToDevice);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&(time_array[0]), start, stop);

	// Invoke kernel
	hipEventRecord(start, 0);

	// ------------------------------------------------------------
	//
	// Invoke kernel function here
	//
	// ------------------------------------------------------------
	threads = 256;	//16;                       //num_points/blocks;
	blocks = (num_points + threads - 1) / threads;//num_points/threads;       //my_log(num_points);
	minimum_distance<<<4, 128>>>(dVx, dVy, dmin_dist, num_points);
	//minimum_distance<<<, BLOCK_SIZE>>>(dVx, dVy, dmin_dist, num_points);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&(time_array[1]), start, stop);

	// Copy result from device memory to host memory 
	hipEventRecord(start, 0);

	hipMemcpy(hmin_dist, dmin_dist, size, hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&(time_array[2]), start, stop);

	printf("Number of Points = %d\n", num_points);
	printf("GPU Host-to-device = %f ms \n", time_array[0]);
	printf("GPU execution time = %f ms \n", time_array[1]);
	printf("GPU Device-to-host = %f ms \n", time_array[2]);
	printf("Minimum distance (GPU) = %e\n", hmin_dist[0]);

	// Compute minimum distance on host to check device computation
	gettimeofday(&cpu_start, NULL);
	dx = hVx[1] - hVx[0];
	dy = hVy[1] - hVy[0];
	distance = sqrtf(dx * dx + dy * dy);
	for (i = 0; i < num_points; i++) {
		for (j = i + 1; j < num_points; j++) {
			dx = hVx[j] - hVx[i];
			dy = hVy[j] - hVy[i];
			Dij = sqrtf(dx * dx + dy * dy);
			if (distance > Dij)
				distance = Dij;
		}
	}
	gettimeofday(&cpu_stop, NULL);
	time_array[3] = 1000 * (cpu_stop.tv_sec - cpu_start.tv_sec)
			+ 0.000001 * (cpu_stop.tv_usec - cpu_start.tv_usec);
	printf("CPU execution time = %f ms\n", time_array[3]);
	printf("Minimum distance (CPU) = %e\n", distance);

	// Free device memory 
	hipFree(dVx);
	hipFree(dVy);
	hipFree(dmin_dist);

	// Free host memory 
	free(hVx);
	free(hVy);
	free(hmin_dist);

	return 0;
}
