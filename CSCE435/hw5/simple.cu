#include "hip/hip_runtime.h"
/*
#include "cuPrintf.cu"
#include <stdio.h>
__global__ void testKernel(int val)
{
  cuPrintf("Value is: %d\n", val);
  printf("output");
}

int main()
{

 // cudaPrintfInit();
  testKernel<<< 2, 3 >>>(10);
  hipDeviceSynchronize();
  //cudaPrintfDisplay(stdout, true);
  //cudaPrintfEnd();
  return 0;
}
*/
/*
#include "stdio.h"

// printf() is only supported
// for devices of compute capability 2.0 and higher
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
  # error printf is only supported on devices of compute capability 2.0 and higher, please compile with -arch=sm_20 or higher    
#endif

__global__ void helloCUDA(float f)
{
    printf("Hello thread %d, f=%f\n", threadIdx.x, f);
}

int main()
{
    helloCUDA<<<1, 5>>>(1.2345f);
    hipDeviceSynchronize();
    return 0;
}

*/

// sample code for the usage of DLog

#include <hip/hip_runtime.h>

#include <iostream>

using namespace std;

#include "DLog.h"

const int numThreads=2;
const int numBlocks=2;

__global__ void foo(){
		DLog<<threadIdx.x;
}

int main(){
		DLog_Init(1024*1024);
		foo<<<numBlocks, numThreads>>>();
		DLog_Dump();
		return 0;
}
